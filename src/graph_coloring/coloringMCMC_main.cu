#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include "coloringMCMC.h"

template<typename nodeW, typename edgeW>
ColoringMCMC<nodeW, edgeW>::ColoringMCMC(Graph<nodeW, edgeW> * inGraph_d, hiprandState * randStates, ColoringMCMCParams param) :
	Colorer<nodeW, edgeW>(inGraph_d),
	graphStruct_d(inGraph_d->getStruct()),
	nnodes(inGraph_d->getStruct()->nNodes),
	prob(inGraph_d->prob),
	randStates(randStates),
	numOfColors(0),
	threadId(0),
	param(param) {

	// configuro la griglia e i blocchi
	numThreads = 32;
	threadsPerBlock = dim3(numThreads, 1, 1);
	blocksPerGrid = dim3((nnodes + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);
	blocksPerGrid_nCol = dim3((param.nCol + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);
	blocksPerGrid_half = dim3(((nnodes / 2) + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

	__customPrintConstructor0_start();

	cuSts = hipMalloc((void**)&coloring_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMalloc((void**)&starColoring_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef TABOO
	cuSts = hipMalloc((void**)&taboo_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // TABOO

	q_h = (float *)malloc(nnodes * sizeof(float));
	cuSts = hipMalloc((void**)&q_d, nnodes * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
	qStar_h = (float *)malloc(nnodes * sizeof(float));
	cuSts = hipMalloc((void**)&qStar_d, nnodes * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);

	conflictCounter_h = (uint32_t *)malloc(nnodes * sizeof(uint32_t));
	cuSts = hipMalloc((void**)&conflictCounter_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMalloc((void**)&colorsChecker_d, nnodes * param.nCol * sizeof(bool));	cudaCheck(cuSts, __FILE__, __LINE__);
#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE)
	cuSts = hipMalloc((void**)&probDistributionLine_d, param.nCol * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE
#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP) || defined(COLOR_BALANCE_EXP)
	cuSts = hipMalloc((void**)&probDistributionExp_d, param.nCol * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_EXP_INIT || COLOR_DECREASE_EXP || COLOR_BALANCE_EXP

#if defined(COLOR_BALANCE_EXP) || defined(TAIL_CUTTING)
	orderedIndex_h = (uint32_t *)malloc(param.nCol * sizeof(uint32_t));
	cuSts = hipMalloc((void**)&orderedIndex_d, param.nCol * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // COLOR_BALANCE_EXP || TAIL_CUTTING


#ifdef STATS
	coloring_h = (uint32_t *)malloc(nnodes * sizeof(uint32_t));
	statsColors_h = conflictCounter_h;
	statsFreeColors_d = conflictCounter_d;
#endif

	__customPrintConstructor1_end();
}

template<typename nodeW, typename edgeW>
ColoringMCMC<nodeW, edgeW>::~ColoringMCMC() {
	cuSts = hipFree(coloring_d); 					cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(starColoring_d); 				cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef TABOO
	cuSts = hipFree(taboo_d); 						cudaCheck(cuSts, __FILE__, __LINE__);
#endif // TABOO

	cuSts = hipFree(colorsChecker_d); 				cudaCheck(cuSts, __FILE__, __LINE__);
#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE)
	cuSts = hipFree(probDistributionLine_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE
#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP) || defined(COLOR_BALANCE_EXP)
	cuSts = hipFree(probDistributionExp_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_EXP_INIT || COLOR_DECREASE_EXP || COLOR_BALANCE_EXP

#if defined(COLOR_BALANCE_EXP) || defined(TAIL_CUTTING)
	free(orderedIndex_h);
	cuSts = hipFree(orderedIndex_d); 			cudaCheck(cuSts, __FILE__, __LINE__);
#endif // COLOR_BALANCE_EXP || TAIL_CUTTING

	cuSts = hipFree(conflictCounter_d); 			cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(q_d); 							cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(qStar_d);						cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef STATS
	free(coloring_h);
#endif

	free(conflictCounter_h);
	free(q_h);
	free(qStar_h);
}

/**
 * Start the coloring on the graph
 */
template<typename nodeW, typename edgeW>
void ColoringMCMC<nodeW, edgeW>::run(int iteration) {

	rip = 0;

	__customPrintRun0_start(iteration);

	cuSts = hipMemset(coloring_d, 0, nnodes * sizeof(uint32_t)); cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef TABOO
	cuSts = hipMemset(taboo_d, 0, nnodes * sizeof(uint32_t)); cudaCheck(cuSts, __FILE__, __LINE__);
#endif // TABOO

#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE) || defined(COLOR_BALANCE_LINE)
	float denomL = 0;
	for (int i = 0; i < param.nCol; i++)
	{
		denomL += exp(-param.lambda * i);
	}
	ColoringMCMC_k::initDistributionLine << < blocksPerGrid_nCol, threadsPerBlock >> > (param.nCol, denomL, param.lambda, probDistributionLine_d);
	hipDeviceSynchronize();
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE || COLOR_BALANCE_LINE

#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP) || defined(COLOR_BALANCE_EXP)
	float denomE = 0;
	for (int i = 0; i < param.nCol; i++)
	{
		denomE += exp(-param.lambda * i);
	}
	ColoringMCMC_k::initDistributionExp << < blocksPerGrid_nCol, threadsPerBlock >> > (param.nCol, denomE, param.lambda, probDistributionExp_d);
	hipDeviceSynchronize();
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE || COLOR_BALANCE_EXP

#ifdef STANDARD_INIT
	ColoringMCMC_k::initColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, randStates);
#endif // STANDARD_INIT

#ifdef DISTRIBUTION_LINE_INIT
	ColoringMCMC_k::initColoringWithDistribution << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, probDistributionLine_d, randStates);
#endif // DISTRIBUTION_LINE_INIT

#ifdef DISTRIBUTION_EXP_INIT
	ColoringMCMC_k::initColoringWithDistribution << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, probDistributionExp_d, randStates);
#endif // DISTRIBUTION_EXP_INIT
	hipDeviceSynchronize();

	__customPrintRun1_init();

	start = std::clock();

	do {

		rip++;

		calcConflicts(conflictCounter, coloring_d);

#if !defined(TAIL_CUTTING)
		if (conflictCounter == 0)
			break;
#else
		if (conflictCounter < 200)
			break;
#endif // TAIL_CUTTING


		__customPrintRun2_conflicts();

		hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));

#ifdef STANDARD
		ColoringMCMC_k::selectStarColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, randStates, param.epsilon, statsFreeColors_d);
		hipDeviceSynchronize();
#endif // STANDARD

#ifdef COLOR_DECREASE_LINE
		ColoringMCMC_k::selectStarColoringDecrease << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, probDistributionLine_d, randStates, param.lambda, param.epsilon, statsFreeColors_d);
#endif // COLOR_DECREASE_LINE

#ifdef COLOR_DECREASE_EXP
		ColoringMCMC_k::selectStarColoringDecrease << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, probDistributionExp_d, randStates, param.lambda, param.epsilon, statsFreeColors_d);
#endif // COLOR_DECREASE_EXP

#ifdef COLOR_BALANCE_LINE
		cuSts = hipMemcpy(coloring_h, coloring_d, nnodes * sizeof(uint32_t), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		memset(statsColors_h, 0, nnodes * sizeof(uint32_t));
		for (int i = 0; i < nnodes; i++) statsColors_h[coloring_h[i]]++;
		for (uint32_t i = 0; i < param.nCol; i++) orderedIndex_h[i] = i;
		std::sort(&orderedIndex_h[0], &orderedIndex_h[param.nCol], [&](int i, int j) {return statsColors_h[i] < statsColors_h[j]; });
		cuSts = hipMemcpy(orderedIndex_d, orderedIndex_h, param.nCol * sizeof(uint32_t), hipMemcpyHostToDevice); cudaCheck(cuSts, __FILE__, __LINE__);

		ColoringMCMC_k::selectStarColoringBalance << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, probDistributionLine_d, orderedIndex_d, randStates, param.lambda, param.epsilon, statsFreeColors_d);
#endif // COLOR_BALANCE_EXP

#ifdef COLOR_BALANCE_EXP
		cuSts = hipMemcpy(coloring_h, coloring_d, nnodes * sizeof(uint32_t), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		memset(statsColors_h, 0, nnodes * sizeof(uint32_t));
		for (int i = 0; i < nnodes; i++) statsColors_h[coloring_h[i]]++;
		for (uint32_t i = 0; i < param.nCol; i++) orderedIndex_h[i] = i;
		std::sort(&orderedIndex_h[0], &orderedIndex_h[param.nCol], [&](int i, int j) {return statsColors_h[i] < statsColors_h[j]; });
		cuSts = hipMemcpy(orderedIndex_d, orderedIndex_h, param.nCol * sizeof(uint32_t), hipMemcpyHostToDevice); cudaCheck(cuSts, __FILE__, __LINE__);

		ColoringMCMC_k::selectStarColoringBalance << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, taboo_d, param.tabooIteration, probDistributionExp_d, orderedIndex_d, randStates, param.lambda, param.epsilon, statsFreeColors_d);
#endif // COLOR_BALANCE_EXP

		hipDeviceSynchronize();

		//hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));
		//ColoringMCMC_k::lookOldColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, q_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, param.epsilon);
		//hipDeviceSynchronize();

		calcConflicts(conflictCounterStar, starColoring_d);

		__customPrintRun3_newConflicts();

		__customPrintRun4();

#ifdef HASTINGS
		calcProbs();

		//param.lambda = -numberOfChangeColorStar * log(param.epsilon); numberOfChangeColorStar cos'�? qualche tentativo vecchio di definire lambda?

		result = param.lambda * (conflictCounter - conflictCounterStar) + p - pStar;
		result = exp(result);

		random = ((float)rand() / (float)RAND_MAX);

		__customPrintRun5();

		//if (random < result) {
		__customPrintRun6_change();
#endif //HASTINGS

		switchPointer = coloring_d;
		coloring_d = starColoring_d;
		starColoring_d = switchPointer;
		//}

		//getStatsNumColors("running_");

	} while (rip < param.maxRip);

#if defined(TAIL_CUTTING)
	cuSts = hipMemcpy(coloring_h, coloring_d, nnodes * sizeof(uint32_t), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
	memset(statsColors_h, 0, nnodes * sizeof(uint32_t));
	for (int i = 0; i < nnodes; i++) statsColors_h[coloring_h[i]]++;
	for (uint32_t i = 0; i < param.nCol; i++) orderedIndex_h[i] = i;
	std::sort(&orderedIndex_h[0], &orderedIndex_h[param.nCol], [&](int i, int j) {return statsColors_h[i] < statsColors_h[j]; });
	cuSts = hipMemcpy(orderedIndex_d, orderedIndex_h, param.nCol * sizeof(uint32_t), hipMemcpyHostToDevice); cudaCheck(cuSts, __FILE__, __LINE__);

	while (conflictCounter > 0) {
		std::cout << "TAGLIO" << std::endl;
		__customPrintRun2_conflicts();
		// set conflictCounter_d vector with 1 or 0 to indicate conflicts
		ColoringMCMC_k::conflictCounter << < blocksPerGrid, threadsPerBlock >> > (nnodes, conflictCounter_d, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs);
		// set colorsChecker_d vector values to 0
		hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));
		// resolve conflicts
		ColoringMCMC_k::tailCutting << < 1, 1 >> > (nnodes, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, conflictCounter, conflictCounter_d, orderedIndex_d);
		calcConflicts(conflictCounter, coloring_d);
		__customPrintRun3_newConflicts();
	}
#endif // TAIL_CUTTING

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;

	if (rip == param.maxRip)
		maxIterReached = true;

	__customPrintRun7_end();
}

//// Questo serve per mantenere le dechiarazioni e definizioni in classi separate
//// E' necessario aggiungere ogni nuova dichiarazione per ogni nuova classe tipizzata usata nel main
template class ColoringMCMC<col, col>;
template class ColoringMCMC<float, float>;

// Original Prob Calc
/*
cuSts = hipMemcpy(qStar_h, qStar_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
cuSts = hipMemcpy(q_h, q_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

pStar = 0;
p = 0;
for (int i = 0; i < nnodes; i++)
{
	pStar += log(qStar_h[i]);
	p += log(q_h[i]);
}

std::cout << "q star: " << pStar << " old:" << p << std::endl;
*/
