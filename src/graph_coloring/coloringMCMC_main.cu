#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include "coloringMCMC.h"

template<typename nodeW, typename edgeW>
ColoringMCMC<nodeW, edgeW>::ColoringMCMC(Graph<nodeW, edgeW> * inGraph_d, hiprandState * randStates, ColoringMCMCParams param) :
	Colorer<nodeW, edgeW>(inGraph_d),
	graphStruct_d(inGraph_d->getStruct()),
	nnodes(inGraph_d->getStruct()->nNodes),
	prob(inGraph_d->prob),
	randStates(randStates),
	numOfColors(0),
	threadId(0),
	param(param) {

	// configuro la griglia e i blocchi
	numThreads = 32;
	threadsPerBlock = dim3(numThreads, 1, 1);
	blocksPerGrid = dim3((nnodes + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);
	blocksPerGrid_nCol = dim3((param.nCol + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);
	blocksPerGrid_half = dim3(((nnodes / 2) + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

	__customPrintConstructor0_start();

	cuSts = hipMalloc((void**)&coloring_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMalloc((void**)&starColoring_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);

	q_h = (float *)malloc(nnodes * sizeof(float));
	cuSts = hipMalloc((void**)&q_d, nnodes * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
	qStar_h = (float *)malloc(nnodes * sizeof(float));
	cuSts = hipMalloc((void**)&qStar_d, nnodes * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);

	conflictCounter_h = (uint32_t *)malloc(nnodes * sizeof(uint32_t));
	cuSts = hipMalloc((void**)&conflictCounter_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMalloc((void**)&colorsChecker_d, nnodes * param.nCol * sizeof(bool));	cudaCheck(cuSts, __FILE__, __LINE__);
#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE)
	cuSts = hipMalloc((void**)&probDistributionLine_d, param.nCol * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE
#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP)
	cuSts = hipMalloc((void**)&probDistributionExp_d, param.nCol * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_EXP_INIT || COLOR_DECREASE_EXP


#ifdef STATS
	coloring_h = (uint32_t *)malloc(nnodes * sizeof(uint32_t));
	statsColors_h = conflictCounter_h;
	statsFreeColors_d = conflictCounter_d;
#endif

	__customPrintConstructor1_end();
}

template<typename nodeW, typename edgeW>
ColoringMCMC<nodeW, edgeW>::~ColoringMCMC() {
	cuSts = hipFree(coloring_d); 					cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(starColoring_d); 				cudaCheck(cuSts, __FILE__, __LINE__);

	cuSts = hipFree(colorsChecker_d); 				cudaCheck(cuSts, __FILE__, __LINE__);
#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE)
	cuSts = hipFree(probDistributionLine_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE
#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP)
	cuSts = hipFree(probDistributionExp_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
#endif // DISTRIBUTION_EXP_INIT || COLOR_DECREASE_EXP

	cuSts = hipFree(conflictCounter_d); 			cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(q_d); 							cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(qStar_d);						cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef STATS
	free(coloring_h);
#endif

	free(conflictCounter_h);
	free(q_h);
	free(qStar_h);
}

/**
 * Start the coloring on the graph
 */
template<typename nodeW, typename edgeW>
void ColoringMCMC<nodeW, edgeW>::run(int iteration) {

	rip = 0;

	__customPrintRun0_start(iteration);

	cuSts = hipMemset(coloring_d, 0, nnodes * sizeof(uint32_t)); cudaCheck(cuSts, __FILE__, __LINE__);

#if defined(DISTRIBUTION_LINE_INIT) || defined(COLOR_DECREASE_LINE)
#ifdef FIXED_N_COLORS
	float denomL = 0;
	for (int i = 0; i < param.nCol; i++)
	{
		denomL += exp(-param.lambda * i);
	}
	ColoringMCMC_k::initDistributionLine << < blocksPerGrid_nCol, threadsPerBlock >> > (param.nCol, denomL, param.lambda, probDistributionLine_d);
#endif // FIXED_N_COLORS
#ifdef DYNAMIC_N_COLORS
	float denomL = 0;
	for (int i = 0; i < param.startingNCol; i++)
	{
		denomL += exp(-param.lambda * i);
	}
	ColoringMCMC_k::initDistributionLine << < blocksPerGrid_nCol, threadsPerBlock >> > (param.startingNCol, denomL, param.lambda, probDistributionLine_d);
#endif // DYNAMIC_N_COLORS

	hipDeviceSynchronize();
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE

#if defined(DISTRIBUTION_EXP_INIT) || defined(COLOR_DECREASE_EXP)
#ifdef FIXED_N_COLORS
	float denomE = 0;
	for (int i = 0; i < param.nCol; i++)
	{
		denomE += exp(-param.lambda * i);
	}
	ColoringMCMC_k::initDistributionExp << < blocksPerGrid_nCol, threadsPerBlock >> > (param.nCol, denomE, param.lambda, probDistributionExp_d);
#endif // FIXED_N_COLORS
#ifdef DYNAMIC_N_COLORS
	float denomE = 0;
	for (int i = 0; i < param.startingNCol; i++)
	{
		denomE += exp(-param.lambda * i);
	}
	ColoringMCMC_k::initDistributionExp << < blocksPerGrid_nCol, threadsPerBlock >> > (param.startingNCol, denomE, param.lambda, probDistributionExp_d);
#endif // DYNAMIC_N_COLORS

	hipDeviceSynchronize();
#endif // DISTRIBUTION_LINE_INIT || COLOR_DECREASE_LINE

#ifdef STANDARD_INIT
#ifdef FIXED_N_COLORS
	ColoringMCMC_k::initColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, randStates);
#endif // FIXED_N_COLORS
#ifdef DYNAMIC_N_COLORS
	ColoringMCMC_k::initColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.startingNCol, randStates);
#endif // DYNAMIC_N_COLORS
#endif // STANDARD_INIT

#ifdef DISTRIBUTION_LINE_INIT
#ifdef FIXED_N_COLORS
	ColoringMCMC_k::initColoringWithDistribution << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, probDistributionLine_d, randStates);
#endif // FIXED_N_COLORS
#ifdef DYNAMIC_N_COLORS
	ColoringMCMC_k::initColoringWithDistribution << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.startingNCol, probDistributionLine_d, randStates);
#endif // DYNAMIC_N_COLORS
#endif // DISTRIBUTION_LINE_INIT

#ifdef DISTRIBUTION_EXP_INIT
#ifdef FIXED_N_COLORS
	ColoringMCMC_k::initColoringWithDistribution << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.nCol, probDistributionExp_d, randStates);
#endif // FIXED_N_COLORS
#ifdef DYNAMIC_N_COLORS
	ColoringMCMC_k::initColoringWithDistribution << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, param.startingNCol, probDistributionExp_d, randStates);
#endif // DYNAMIC_N_COLORS
#endif // DISTRIBUTION_EXP_INIT
	hipDeviceSynchronize();

	__customPrintRun1_init();

	start = std::clock();

	do {

		rip++;

		calcConflicts(conflictCounter, coloring_d);

		if (conflictCounter == 0)
			break;

		__customPrintRun2_conflicts();

		hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));

#ifdef STANDARD
#ifdef FIXED_N_COLORS
		ColoringMCMC_k::selectStarColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, randStates, param.epsilon, statsFreeColors_d);
		hipDeviceSynchronize();
#endif // FIXED_N_COLORS
#ifdef DYNAMIC_N_COLORS
		ColoringMCMC_k::selectStarColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.startingNCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, randStates, param.epsilon, statsFreeColors_d);
		cuSts = hipMemcpy(qStar_h, qStar_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		for (uint32_t i = 0; i < nnodes && param.startingNCol < param.nCol; i++)
		{
			//if (coloring_h[i] == param.startingNCol)
			if (qStar_h[i] == 1)
			{
				//param.startingNCol++;
				param.startingNCol += 1;
				i = nnodes;
			}

		}
		std::cout << "startingNCol = " << param.startingNCol << std::endl;
#endif // DYNAMIC_N_COLORS
#endif // STANDARD

#ifdef COLOR_DECREASE_LINE
#ifdef FIXED_N_COLORS
		ColoringMCMC_k::selectStarColoringDecrease << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, probDistributionLine_d, randStates, param.epsilon, statsFreeColors_d);
#endif // FIXED_N_COLORS
#ifdef DYNAMIC_N_COLORS
		ColoringMCMC_k::selectStarColoringDecrease << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.startingNCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, probDistributionLine_d, randStates, param.epsilon, statsFreeColors_d);
		cuSts = hipMemcpy(qStar_h, qStar_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		for (uint32_t i = 0; i < nnodes && param.startingNCol < param.nCol; i++)
		{
			//if (coloring_h[i] == param.startingNCol)
			if (qStar_h[i] == 1)
			{
				//param.startingNCol++;
				param.startingNCol += 1;
				float denomL = 0;
				for (int i = 0; i < param.startingNCol; i++)
				{
					denomL += exp(-param.lambda * i);
				}
				ColoringMCMC_k::initDistributionLine << < blocksPerGrid_nCol, threadsPerBlock >> > (param.startingNCol, denomL, param.lambda, probDistributionLine_d);
				i = nnodes;
			}

		}
		std::cout << "startingNCol = " << param.startingNCol << std::endl;
#endif // DYNAMIC_N_COLORS
#endif // COLOR_DECREASE_LINE

#ifdef COLOR_DECREASE_EXP
#ifdef FIXED_N_COLORS
		ColoringMCMC_k::selectStarColoringDecrease << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, probDistributionExp_d, randStates, param.lambda, param.epsilon, statsFreeColors_d);
#endif // FIXED_N_COLORS
#ifdef DYNAMIC_N_COLORS
		ColoringMCMC_k::selectStarColoringDecrease << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, qStar_d, param.startingNCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, probDistributionExp_d, randStates, param.epsilon, statsFreeColors_d);
		cuSts = hipMemcpy(qStar_h, qStar_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		for (uint32_t i = 0; i < nnodes && param.startingNCol < param.nCol; i++)
		{
			//if (coloring_h[i] == param.startingNCol)
			if (qStar_h[i] == 1)
			{
				//param.startingNCol++;
				param.startingNCol += 1;
				float denomE = 0;
				for (int i = 0; i < param.startingNCol; i++)
				{
					denomE += exp(-param.lambda * i);
				}
				ColoringMCMC_k::initDistributionExp << < blocksPerGrid_nCol, threadsPerBlock >> > (param.startingNCol, denomE, param.lambda, probDistributionExp_d);
				i = nnodes;
			}

		}
		std::cout << "startingNCol = " << param.startingNCol << std::endl;
#endif // DYNAMIC_N_COLORS
#endif // COLOR_DECREASE_EXP

		hipDeviceSynchronize();

		hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));
		ColoringMCMC_k::lookOldColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, starColoring_d, q_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, param.epsilon);
		hipDeviceSynchronize();

		calcConflicts(conflictCounterStar, starColoring_d);

		__customPrintRun3_newConflicts();

		__customPrintRun4();

		//calcProbs();

		//param.lambda = -numberOfChangeColorStar * log(param.epsilon);

		//result = param.lambda * (conflictCounter - conflictCounterStar) + p - pStar;
		//result = exp(result);

		//random = ((float)rand() / (float)RAND_MAX);

		__customPrintRun5();

		//if (random < result) {
		__customPrintRun6_change();

		switchPointer = coloring_d;
		coloring_d = starColoring_d;
		starColoring_d = switchPointer;
		//}

		//getStatsNumColors("running_");

	} while (rip < param.maxRip);
	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;

	if (rip == param.maxRip)
		maxIterReached = true;

	__customPrintRun7_end();
}

//// Questo serve per mantenere le dechiarazioni e definizioni in classi separate
//// E' necessario aggiungere ogni nuova dichiarazione per ogni nuova classe tipizzata usata nel main
template class ColoringMCMC<col, col>;
template class ColoringMCMC<float, float>;

// Original Prob Calc
/*
cuSts = hipMemcpy(qStar_h, qStar_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
cuSts = hipMemcpy(q_h, q_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

pStar = 0;
p = 0;
for (int i = 0; i < nnodes; i++)
{
	pStar += log(qStar_h[i]);
	p += log(q_h[i]);
}

std::cout << "q star: " << pStar << " old:" << p << std::endl;
*/
