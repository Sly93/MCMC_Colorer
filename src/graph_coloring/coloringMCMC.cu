#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include "coloringMCMC.h"

#define STATS
#define PRINTS

template<typename nodeW, typename edgeW>
ColoringMCMC<nodeW, edgeW>::ColoringMCMC(Graph<nodeW, edgeW> * inGraph_d, hiprandState * randStates, ColoringMCMCParams params) :
	Colorer<nodeW, edgeW>(inGraph_d),
	graphStruct_d(inGraph_d->getStruct()),
	nnodes(inGraph_d->getStruct()->nNodes),
	nedges(inGraph_d->getStruct()->nCleanEdges),
	randStates(randStates),
	numOfColors(0),
	threadId(0),
	param(params) {

	// configuro la griglia e i blocchi
	numThreads = 32;
	threadsPerBlock = dim3(numThreads, 1, 1);
	blocksPerGrid = dim3((nnodes + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);
	blocksPerGrid_edges = dim3((nedges + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);
	blocksPerGrid_half_edges = dim3(((nedges / 2) + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

	cuSts = hipMalloc((void**)&coloring_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMalloc((void**)&newColoring_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);

	probColoring_h = (float *)malloc(nnodes * sizeof(float));
	cuSts = hipMalloc((void**)&probColoring_d, nnodes * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);
	probNewColoring_h = (float *)malloc(nnodes * sizeof(float));
	cuSts = hipMalloc((void**)&probNewColoring_d, nnodes * sizeof(float));	cudaCheck(cuSts, __FILE__, __LINE__);

	counter_h = (uint32_t *)malloc(nedges * sizeof(uint32_t));
	cuSts = hipMalloc((void**)&counter_d, nedges * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
	newCounter_h = (uint32_t *)malloc(nedges * sizeof(uint32_t));
	cuSts = hipMalloc((void**)&newCounter_d, nedges * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);

	cuSts = hipMalloc((void**)&colorsChecker_d, nnodes * param.nCol * sizeof(bool));	cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipMalloc((void**)&orderedColors_d, nnodes * param.nCol * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);

#ifdef STATS
	statsFreeColors_h = (uint32_t *)malloc(nnodes * sizeof(uint32_t));
	cuSts = hipMalloc((void**)&statsFreeColors_d, nnodes * sizeof(uint32_t));	cudaCheck(cuSts, __FILE__, __LINE__);
#endif

	divider = 1 / ((float)params.nCol);
}

template<typename nodeW, typename edgeW>
ColoringMCMC<nodeW, edgeW>::~ColoringMCMC() {
	cuSts = hipFree(coloring_d); 			cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(newColoring_d); 		cudaCheck(cuSts, __FILE__, __LINE__);

	cuSts = hipFree(colorsChecker_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(orderedColors_d); 		cudaCheck(cuSts, __FILE__, __LINE__);

	cuSts = hipFree(counter_d); 			cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(newCounter_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(probColoring_d); 		cudaCheck(cuSts, __FILE__, __LINE__);
	cuSts = hipFree(probNewColoring_d); 	cudaCheck(cuSts, __FILE__, __LINE__);

	free(counter_h);
	free(newCounter_h);
	free(probColoring_h);
	free(probNewColoring_h);

#ifdef STATS
	free(statsFreeColors_h);
	cuSts = hipFree(statsFreeColors_d);	cudaCheck(cuSts, __FILE__, __LINE__);
#endif
}

/**
* Set coloring_d with random colors
*/
__global__ void ColoringMCMC_k::initColoring(uint32_t nnodes, uint32_t * coloring_d, float divider, hiprandState * states) {

	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= nnodes)
		return;

	float randnum = hiprand_uniform(&states[idx]);

	int color = (int)(randnum / divider);

	coloring_d[idx] = color;
	//coloring_d[idx] = 0;
}

/**
* For all the edges of the graph, set the value of counter_d to 0 or 1 if the nodes of the edge have the same color
*/
__global__ void ColoringMCMC_k::conflictChecker(uint32_t nedges, uint32_t * counter_d, uint32_t * coloring_d, node_sz * edges) {

	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= nedges)
		return;

	uint32_t idx0 = idx * 2;
	uint32_t idx1 = idx0 + 1;

	uint32_t node0 = edges[idx0];
	uint32_t node1 = edges[idx1];

	uint32_t col0 = coloring_d[node0];
	uint32_t col1 = coloring_d[node1];

	counter_d[idx] = col0 == col1;
}

/**
* Parallel sum reduction inside a single warp
*/
//template <uint32_t blockSize>
__device__ void ColoringMCMC_k::warpReduction(volatile int *sdata, uint32_t tid, uint32_t blockSize) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

/*
* Parallel sum reduction inside a block and write the partial result in counter_d.
* At the end, counter_d have n partial results for the first n positions where n is the number of blocks called.

* refs: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
*/
//template <uint32_t blockSize>
__global__ void ColoringMCMC_k::sumReduction(uint32_t nedges, uint32_t * counter_d) {

	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= nedges)
		return;

	extern	__shared__ int sdata[];

	uint32_t tid = threadIdx.x;
	uint32_t blockSize = blockDim.x;
	uint32_t i = (blockSize * 2) * blockIdx.x + tid;

	sdata[tid] = counter_d[i] + counter_d[i + blockSize];

	/*uint32_t gridSize = (blockSize * 2) * gridDim.x;
	sdata[tid] = 0;
	while (i < nedges) {
		sdata[tid] += counter_d[i] + counter_d[i + blockSize];
		i += gridSize;
	}*/
	__syncthreads();

	//useless for blocks of dim <= 64
	if (blockSize >= 512)
	{
		if (tid < 256)
			sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if (blockSize >= 256)
	{
		if (tid < 128)
			sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if (blockSize >= 128)
	{
		if (tid < 64)
			sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}

	if (tid < 32)
		//ColoringMCMC_k::warpReduction<blockSize>(sdata, tid);
		ColoringMCMC_k::warpReduction(sdata, tid, blockSize);

	if (tid == 0)
		counter_d[blockIdx.x] = sdata[0];
}

/**
* For every node, look at neighbors and select a new color.
* This will be write in newColoring_d and the probability of the chosen color will be write in probNewColoring_d
*/
__global__ void ColoringMCMC_k::selectNewColoring(uint32_t nnodes, uint32_t * newColoring_d, float * probNewColoring_d, col_sz nCol, uint32_t * coloring_d, node_sz * cumulDegs, node * neighs, bool * colorsChecker_d, uint32_t * orderedColors_d, hiprandState * states, float epsilon, uint32_t * statsFreeColors_d) {

	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= nnodes)
		return;

	uint32_t index = cumulDegs[idx];							//index of the node in neighs
	uint32_t nneighs = cumulDegs[idx + 1] - index;				//number of neighbors

	uint32_t nodeCol = coloring_d[idx];							//node color

	bool * colorsChecker = &(colorsChecker_d[idx * nCol]);		//array used to set to 1 or 0 the colors occupied from the neighbors
	for (int i = 0; i < nneighs; i++)
		colorsChecker[coloring_d[neighs[index + i]]] = 1;

	uint32_t * orderedColors = &(orderedColors_d[idx * nCol]);	//array containing previously occupied colors and then free ones
	uint32_t Zp = nCol, Zn = 0;									//number of free colors (p) and occupied colors (n)
	for (int i = 0; i < nCol; i++)
	{
		orderedColors[Zn] += i * (1 - (1 - colorsChecker[i]));
		orderedColors[Zp - 1] += i * (1 - colorsChecker[i]);
		Zn += colorsChecker[i];
		Zp -= 1 - colorsChecker[i];
	}
	Zp = nCol - Zn;

#ifdef STATS
	statsFreeColors_d[idx] = Zp;
#endif

	if (!Zp)													//manage exception of no free colors
	{
		newColoring_d[idx] = nodeCol;
		probNewColoring_d[idx] = 1;
		return;
	}

	float randnum = hiprand_uniform(&states[idx]);				//random number

	float threshold;
	uint32_t selectedIndex = 0;									//selected index for orderedColors to select the new color
	if (colorsChecker[nodeCol])									//if node color is used by neighbors
	{
		threshold = 1 - epsilon * Zn;							//threshold used to randomly determine whether to extract a free color or a busy one
		if (randnum < threshold)
		{
			selectedIndex = ((randnum * Zp) / threshold) + Zn;	//get the selected index
			probNewColoring_d[idx] = (1 - epsilon * Zn) / Zp;	//save the probability of the color chosen
		}
		else
		{
			selectedIndex = ((randnum - threshold) * Zn) / (1 - threshold);	//get the selected index
			probNewColoring_d[idx] = epsilon;					//save the probability of the color chosen
		}
		newColoring_d[idx] = orderedColors[selectedIndex];		//save the new color
	}
	else
	{
		threshold = 1 - epsilon * (nCol - 1);					//threshold used to randomly determine whether to extract a occupied color
																//or keep the same
		if (randnum < threshold)
		{
			newColoring_d[idx] = nodeCol;						//keep the same color
			probNewColoring_d[idx] = 1 - ((nCol - 1) * epsilon);//save the probability of the color chosen
		}
		else
		{
			selectedIndex = ((randnum - threshold) * Zn) / (1 - threshold);	//get the selected index
			newColoring_d[idx] = orderedColors[selectedIndex];	//save the new color
			probNewColoring_d[idx] = epsilon;					//save the probability of the color chosen
		}
	}
}

/**
* For every node, look at neighbors.
* The probability of the old color will be write in probColoring_d
*/
__global__ void ColoringMCMC_k::lookOldColoring(uint32_t nnodes, float * probColoring_d, col_sz nCol, uint32_t * newColoring_d, uint32_t * coloring_d, node_sz * cumulDegs, node * neighs, bool * colorsChecker_d, float epsilon) {
	uint32_t idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx >= nnodes)
		return;

	uint32_t index = cumulDegs[idx];							//index of the node in neighs
	uint32_t nneighs = cumulDegs[idx + 1] - index;				//number of neighbors

	uint32_t nodeCol = coloring_d[idx];							//node color
	uint32_t nodeNewCol = newColoring_d[idx];					//node new color

	bool * colorsChecker = &(colorsChecker_d[idx * nCol]);		//array used to set to 1 or 0 the colors occupied from the neighbors
	for (int i = 0; i < nneighs; i++)
		colorsChecker[newColoring_d[neighs[index + i]]] = 1;

	uint32_t Zp = nCol, Zn = 0;									//number of free colors (p) and occupied colors (n)
	for (int i = 0; i < nCol; i++)
		Zn += colorsChecker[i];
	Zp = nCol - Zn;

	if (!Zp)													//manage exception of no free colors
	{
		probColoring_d[idx] = 1;
		return;
	}

	if (colorsChecker[nodeNewCol])								//if node color is used by neighbors
	{
		if (!colorsChecker[nodeCol])
			probColoring_d[idx] = (1 - epsilon * Zn) / Zp;		//save the probability of the old color
		else
			probColoring_d[idx] = epsilon;						//save the probability of the old color
	}
	else
	{
		if (nodeNewCol == nodeCol)
			probColoring_d[idx] = 1 - ((nCol - 1) * epsilon);	//save the probability of the old color
		else
			probColoring_d[idx] = epsilon;						//save the probability of the old color
	}
}

/**
 * Start the coloring on the graph
 */
template<typename nodeW, typename edgeW>
void ColoringMCMC<nodeW, edgeW>::run() {

	cuSts = hipMemset(coloring_d, 0, nnodes * sizeof(uint32_t)); cudaCheck(cuSts, __FILE__, __LINE__);

	ColoringMCMC_k::initColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, coloring_d, divider, randStates);
	hipDeviceSynchronize();

	do {

		rip++;

		// Original
		/*ColoringMCMC_k::conflictChecker << < blocksPerGrid_edges, threadsPerBlock >> > (nedges, counter_d, coloring_d, graphStruct_d->edges);
		hipDeviceSynchronize();

		cuSts = hipMemcpy(counter_h, counter_d, nedges * sizeof(node_sz), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

		counter = 0;
		for (int i = 0; i < nedges; i++)
			counter += counter_h[i];
		*/
		// End Original

		// Parallel
		ColoringMCMC_k::conflictChecker << < blocksPerGrid_edges, threadsPerBlock >> > (nedges, counter_d, coloring_d, graphStruct_d->edges);
		hipDeviceSynchronize();

		// With Template
		//ColoringMCMC_k::sumReduction <32><< < blocksPerGrid_half_edges, threadsPerBlock, threadsPerBlock.x * sizeof(uint32_t) >> > (nedges, counter_d);
		ColoringMCMC_k::sumReduction << < blocksPerGrid_half_edges, threadsPerBlock, threadsPerBlock.x * sizeof(uint32_t) >> > (nedges, counter_d);
		hipDeviceSynchronize();

		cuSts = hipMemcpy(counter_h, counter_d, blocksPerGrid_half_edges.x * sizeof(node_sz), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

		counter = 0;
		for (int i = 0; i < blocksPerGrid_half_edges.x; i++)
			counter += counter_h[i];
		// End Parallel

		if (counter == 0)
			break;

#ifdef PRINTS
		std::cout << "***** Tentativo numero: " << rip << std::endl;
		std::cout << "conflitti rilevati: " << counter << std::endl;
#endif // PRINTS

		hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));
		hipMemset(orderedColors_d, 0, nnodes * param.nCol * sizeof(uint32_t));
		ColoringMCMC_k::selectNewColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, newColoring_d, probNewColoring_d, param.nCol, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, orderedColors_d, randStates, param.epsilon, statsFreeColors_d);
		hipDeviceSynchronize();

		hipMemset(colorsChecker_d, 0, nnodes * param.nCol * sizeof(bool));
		ColoringMCMC_k::lookOldColoring << < blocksPerGrid, threadsPerBlock >> > (nnodes, probColoring_d, param.nCol, newColoring_d, coloring_d, graphStruct_d->cumulDegs, graphStruct_d->neighs, colorsChecker_d, param.epsilon);
		hipDeviceSynchronize();

#ifdef STATS
		cuSts = hipMemcpy(statsFreeColors_h, statsFreeColors_d, nnodes * sizeof(uint32_t), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		statsFreeColors_max = statsFreeColors_avg = 0;
		statsFreeColors_min = param.nCol + 1;
		for (uint32_t i = 0; i < nnodes; i++) {
			uint32_t freeColors = statsFreeColors_h[i];
			statsFreeColors_avg += freeColors;
			statsFreeColors_max = (freeColors > statsFreeColors_max) ? freeColors : statsFreeColors_max;
			statsFreeColors_min = (freeColors < statsFreeColors_min) ? freeColors : statsFreeColors_min;
		}
		statsFreeColors_avg /= (float)nnodes;
		std::cout << "Max Free Colors: " << statsFreeColors_max << " - Min Free Colors: " << statsFreeColors_min << " - AVG Free Colors: " << statsFreeColors_avg << std::endl;
#endif

		cuSts = hipMemcpy(probNewColoring_h, probNewColoring_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);
		cuSts = hipMemcpy(probColoring_h, probColoring_d, nnodes * sizeof(float), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

		probColoring = 1;
		probNewColoring = 1;
		for (int i = 0; i < nnodes; i++)
		{
			probColoring *= probColoring_h[i];
			probNewColoring *= probNewColoring_h[i];
		}

#ifdef PRINTS
		//std::cout << "probColoring: " << probColoring << " probNewColoring: " << probNewColoring << std::endl;
#endif // PRINTS

		// Original
		/*ColoringMCMC_k::conflictChecker << < blocksPerGrid_edges, threadsPerBlock >> > (nedges, newCounter_d, newColoring_d, graphStruct_d->edges);
		hipDeviceSynchronize();

		cuSts = hipMemcpy(newCounter_h, newCounter_d, nedges * sizeof(node_sz), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

		newCounter = 0;
		for (int i = 0; i < nedges; i++)
			newCounter += newCounter_h[i];
		*/
		// End Original

		// Parallel
		ColoringMCMC_k::conflictChecker << < blocksPerGrid_edges, threadsPerBlock >> > (nedges, newCounter_d, newColoring_d, graphStruct_d->edges);
		hipDeviceSynchronize();

		//ColoringMCMC_k::sumReduction <32><< < blocksPerGrid_half_edges, threadsPerBlock, threadsPerBlock.x * sizeof(uint32_t) >> > (nedges, newCounter_d);
		ColoringMCMC_k::sumReduction << < blocksPerGrid_half_edges, threadsPerBlock, threadsPerBlock.x * sizeof(uint32_t) >> > (nedges, newCounter_d);
		hipDeviceSynchronize();

		cuSts = hipMemcpy(newCounter_h, newCounter_d, blocksPerGrid_half_edges.x * sizeof(node_sz), hipMemcpyDeviceToHost); cudaCheck(cuSts, __FILE__, __LINE__);

		newCounter = 0;
		for (int i = 0; i < blocksPerGrid_half_edges.x; i++)
			newCounter += newCounter_h[i];
		// End Parallel

#ifdef PRINTS
		std::cout << "nuovi conflitti rilevati: " << newCounter << std::endl;
#endif // PRINTS

		result = exp(-param.lambda * ((int64_t)newCounter - (int64_t)counter));		//exp(n) = e ^ (n)

#ifdef PRINTS
		//std::cout << "result:" << result << " lambda:" << param.lambda << " newCounter:" << newCounter << " counter:" << counter << std::endl;
#endif // PRINTS 

		result = (result * probNewColoring) / probColoring;

#ifdef PRINTS
		//std::cout << "result: " << result << std::endl;
#endif // PRINTS

		result = result > 1 ? 1 : result;

#ifdef PRINTS
		//std::cout << "result: " << result << std::endl;
#endif // PRINTS

		//********************************************************************** LOG
		//probColoring = 0;
		//probNewColoring = 0;
		//for (int i = 0; i < nnodes; i++)
		//{
			//probColoring += log(probColoring_h[i]);
			//probNewColoring += log(probNewColoring_h[i]);
		//}
		//result = -param.lambda * ((int64_t)newCounter - (int64_t)counter) + probColoring - probNewColoring;
		//result = exp(result);

#ifdef PRINTS
		//std::cout << "result log: " << result << std::endl;
#endif // PRINTS

		//**********************************************************************

		random = ((float)rand() / (float)RAND_MAX);

#ifdef PRINTS
		//std::cout << "random: " << random << std::endl;
#endif // PRINTS

		//if (random < result) {
#ifdef PRINTS
		std::cout << "CHANGE" << std::endl;
#endif // PRINTS
		temp = coloring_d;
		coloring_d = newColoring_d;
		newColoring_d = temp;
		//}

	} while (rip < param.maxRip);

}

//// Questo serve per mantenere le dechiarazioni e definizioni in classi separate
//// E' necessario aggiungere ogni nuova dichiarazione per ogni nuova classe tipizzata usata nel main
template class ColoringMCMC<col, col>;
template class ColoringMCMC<float, float>;
