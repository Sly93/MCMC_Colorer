#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include <iostream>
#include <ctime>
#include "utils/ArgHandle.h"
#include "utils/fileImporter.h"
#include "utils/dbg.h"
#include "graph/graph.h"
#include "graph/graphCPU.cpp"
#include "graph/graphGPU.cu"
#include "graph_coloring/coloring.h"
#include "graph_coloring/coloringMCMC_CPU.h"
#include "graph_coloring/coloringLuby.h"
#include "graph_coloring/coloringMCMC.h"
#include "GPUutils/GPURandomizer.h"
#include "easyloggingpp/easyloggingpp/easylogging++.h"

#define INLINE_ARGS
//argomenti tipo : --data net.txt --label lab.txt --gene gene.txt

bool		g_traceLogEn;	// Declared in utils/miscUtils.h
dbg		*	g_debugger;

INITIALIZE_EASYLOGGINGPP

void newComponents(Graph<float, float> * test) {

	// arches rappresenta una lista di coppie di nodi che consistono nei singoli archi del grafo (senza ripetizioni)
	std::vector<node_sz> tempEdges;

	for (int i = 0; i < test->getStruct()->nNodes; i++)
	{
		int index = test->getStruct()->cumulDegs[i];
		int numOfNeighbors = test->getStruct()->cumulDegs[i + 1] - index;

		for (int j = 0; j < numOfNeighbors; j++)
		{
			if (i < test->getStruct()->neighs[index + j]) {
				tempEdges.push_back(i);
				tempEdges.push_back(test->getStruct()->neighs[index + j]);
			}
		}
	}

	test->getStruct()->nCleanEdges = tempEdges.size() / 2;
	test->getStruct()->edges = new node_sz[tempEdges.size()];
	for (int i = 0; i < tempEdges.size(); i++)
	{
		test->getStruct()->edges[i] = tempEdges[i];
	}

	std::cout << "Archi singoli: " << (tempEdges.size() / 2) << std::endl;

	//node_sz	* edges = &tempEdges[0];
	//test->getStruct()->edges = edges;
	//test->getStruct()->edges = tempEdges.data();
}

int main(int argc, char *argv[]) {

	////EasyLogging++
	START_EASYLOGGINGPP(argc, argv);
	el::Configurations conf("../src/logger.conf");
	el::Loggers::reconfigureLogger("default", conf);
	el::Loggers::reconfigureAllLoggers(conf);

	el::Configuration * loggerConf = conf.get(el::Level::Trace, el::ConfigurationType::Enabled);
	g_traceLogEn = (loggerConf->value() == "true");
	// Debugger pre-init
	g_debugger = nullptr;

#ifdef INLINE_ARGS
	argc = 7;
	argv[1] = "--data";
	argv[2] = "net.txt";
	argv[3] = "--label";
	argv[4] = "lab.txt";
	argv[5] = "--gene";
	argv[6] = "gene.txt";
#endif // INLINE_ARGS

	// Commandline arguments
	ArgHandle commandLine(argc, argv);
	commandLine.processCommandLine();

	//uint32_t			N				= commandLine.n;
	//uint32_t			M				= commandLine.m;
	//float				prob			= (float) commandLine.prob;
	uint32_t			seed = commandLine.seed;
	std::string			graphFileName = commandLine.dataFilename;
	std::string			labelsFileName = commandLine.labelFilename;

	//seed = 10000;

	std::clock_t start;
	double duration;

	bool GPUEnabled = 1;

	fileImporter fImport(graphFileName, labelsFileName);
	Graph<float, float> test(&fImport, !GPUEnabled);
	LOG(TRACE) << "Nodi: " << test.getStruct()->nNodes << " - Archi: " << test.getStruct()->nEdges;
	LOG(TRACE) << "minDeg: " << test.getMinNodeDeg() << " - maxDeg: " << test.getMaxNodeDeg() << " - meanDeg: "
		<< test.getMeanNodeDeg();

	newComponents(&test);

	//// CPU greedy coloring
	//// Don't know if this still works...
	// Graph<col, col> graph( N, GPUEnabled );  	// random graph
	// ColoringGeedyCPU<col,col> colGreedyCPU(&graph);
	// colGreedyCPU.run();
	// cout << "Greedy-CPU coloring elapsed time: " << colGreedyCPU.getElapsedTime() << "(sec)" << endl;
	//colGreedyCPU.print(0);

	Graph<float, float> graph_d(&test);
	//// GPU Luby coloring
	GPURand GPURandGen(test.getStruct()->nNodes, (long)commandLine.seed);

	/*ColoringLuby<float, float> colLuby(&graph_d, GPURandGen.randStates);
	start = std::clock();
	colLuby.run_fast();
	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	LOG(TRACE) << TXT_BIYLW << "LubyGPU - number of colors: " << colLuby.getColoringGPU()->nCol << TXT_NORML;
	LOG(TRACE) << TXT_BIYLW << "LubyGPU elapsed time: " << duration << TXT_NORML;*/

	ColoringMCMCParams params;
	params.nCol = 80;	//test.getMaxNodeDeg() / 2.0f;
	params.epsilon = 1e-8f;
	params.lambda = 0.002f;
	params.ratioFreezed = 1e-2;
	params.maxRip = 250;

	ColoringMCMC_CPU<float, float> mcmc_cpu(&test, params, seed);
	g_debugger = new dbg(&test, &mcmc_cpu);
	start = std::clock();
	mcmc_cpu.run();
	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;

	LOG(TRACE) << TXT_BIYLW << "MCMC_CPU elapsed time: " << duration << TXT_NORML;

	ColoringMCMC<float, float> colMCMC(&graph_d, GPURandGen.randStates, params);

	start = std::clock();
	colMCMC.run();
	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;

	LOG(TRACE) << TXT_BIYLW << "Elapsed time: " << duration << TXT_NORML;

	if (g_debugger != nullptr)
		delete g_debugger;

	return EXIT_SUCCESS;
}
