#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include <iostream>
#include <ctime>
#include "utils/ArgHandle.h"
#include "utils/fileImporter.h"
#include "graph/graph.h"
#include "graph/graphCPU.cpp"
#include "graph/graphGPU.cu"
#include "graph_coloring/coloring.h"
#include "graph_coloring/coloringMCMC_CPU.h"
#include "graph_coloring/coloringLuby.h"
#include "graph_coloring/coloringMCMC.h"
#include "GPUutils/GPURandomizer.h"
#include "easyloggingpp/easylogging++.h"

INITIALIZE_EASYLOGGINGPP

int main(int argc, char *argv[]) {

	////EasyLogging++
	START_EASYLOGGINGPP(argc, argv);
    el::Configurations conf("logger.conf");
    el::Loggers::reconfigureLogger("default", conf);
    el::Loggers::reconfigureAllLoggers(conf);
	// Commandline arguments
	ArgHandle commandLine( argc, argv );
	commandLine.processCommandLine();

	//uint32_t			N				= commandLine.n;
	//uint32_t			M				= commandLine.m;
	//float				prob			= (float) commandLine.prob;
	uint32_t			seed			= commandLine.seed;
	std::string			graphFileName	= commandLine.dataFilename;
	std::string			labelsFileName	= commandLine.labelFilename;

	std::clock_t start;
	double duration;

	bool GPUEnabled = 1;

	fileImporter fImport( graphFileName, labelsFileName );
	Graph<float, float> test( &fImport, !GPUEnabled );
	std::cout << "Nodi: " << test.getStruct()->nNodes << " - Archi: " << test.getStruct()->nEdges << std::endl;

	//// CPU greedy coloring
	// Graph<col, col> graph( N, GPUEnabled );  	// random graph
	// ColoringGeedyCPU<col,col> colGreedyCPU(&graph);
	// colGreedyCPU.run();
	// cout << "Greedy-CPU coloring elapsed time: " << colGreedyCPU.getElapsedTime() << "(sec)" << endl;
	//colGreedyCPU.print(0);

	Graph<float, float> graph_d( &test );
	//// GPU Luby coloring
	// ColoringLuby<float, float> colLuby(&graph_d, GPURandGen.randStates);
	// colLuby.run_fast();

	ColoringMCMCParams aa;
	aa.nCol = 3;
	aa.epsilon = 1e-3;
	aa.lambda = 2.0f;
	aa.ratioFreezed = 1e-2;

	//// GPU MCMC coloring
	// GPURand GPURandGen( test.getStruct()->nNodes, (long) commandLine.seed );
	// ColoringMCMC<float, float> colMCMC(&graph_d, GPURandGen.randStates, aa);
	// start = std::clock();
	// colMCMC.run();
	// duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

	ColoringMCMC_CPU<float, float> mcmc_cpu( &test, aa, seed );
	start = std::clock();
	mcmc_cpu.run();
	duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

	std::cout << "Elapsed time: " << duration << std::endl;

	return EXIT_SUCCESS;
}
