#include "hip/hip_runtime.h"
// This is a personal academic project. Dear PVS-Studio, please check it.
// PVS-Studio Static Code Analyzer for C, C++ and C#: http://www.viva64.com
#include <iostream>
#include <ctime>
#include "utils/ArgHandle.h"
#include "utils/fileImporter.h"
#include "utils/dbg.h"
#include "graph/graph.h"
#include "graph/graphCPU.cpp"
#include "graph/graphGPU.cu"
#include "graph_coloring/coloring.h"
#include "graph_coloring/coloringMCMC_CPU.h"
#include "graph_coloring/coloringLuby.h"
#include "graph_coloring/coloringMCMC.h"
#include "GPUutils/GPURandomizer.h"
#include "easyloggingpp/easylogging++.h"

bool		g_traceLogEn;	// Declared in utils/miscUtils.h
dbg		*	g_debugger;

INITIALIZE_EASYLOGGINGPP

int main(int argc, char *argv[]) {

	////EasyLogging++
	START_EASYLOGGINGPP(argc, argv);
    el::Configurations conf("../logger.conf");
    el::Loggers::reconfigureLogger("default", conf);
    el::Loggers::reconfigureAllLoggers(conf);

	el::Configuration * loggerConf = conf.get( el::Level::Trace, el::ConfigurationType::Enabled );
	g_traceLogEn = (loggerConf->value() == "true");
	// Debugger pre-init
	g_debugger = nullptr;

	// Commandline arguments
	ArgHandle commandLine( argc, argv );
	commandLine.processCommandLine();

	//uint32_t			N				= commandLine.n;
	//uint32_t			M				= commandLine.m;
	//float				prob			= (float) commandLine.prob;
	uint32_t			seed			= commandLine.seed;
	std::string			graphFileName	= commandLine.dataFilename;
	std::string			labelsFileName	= commandLine.labelFilename;

	//seed = 10000;

	std::clock_t start;
	double duration;

	bool GPUEnabled = 1;

	fileImporter fImport( graphFileName, labelsFileName );
	Graph<float, float> test( &fImport, !GPUEnabled );
	LOG(TRACE) << "Nodi: " << test.getStruct()->nNodes << " - Archi: " << test.getStruct()->nEdges;
	LOG(TRACE) << "minDeg: " << test.getMinNodeDeg() << " - maxDeg: " << test.getMaxNodeDeg() << " - meanDeg: "
		<< test.getMeanNodeDeg();

	//// CPU greedy coloring
	// Graph<col, col> graph( N, GPUEnabled );  	// random graph
	// ColoringGeedyCPU<col,col> colGreedyCPU(&graph);
	// colGreedyCPU.run();
	// cout << "Greedy-CPU coloring elapsed time: " << colGreedyCPU.getElapsedTime() << "(sec)" << endl;
	//colGreedyCPU.print(0);

	Graph<float, float> graph_d( &test );
	//// GPU Luby coloring
	GPURand GPURandGen( test.getStruct()->nNodes, (long) commandLine.seed );
	ColoringLuby<float, float> colLuby(&graph_d, GPURandGen.randStates);
	start = std::clock();
	colLuby.run_fast();
	duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
	LOG(TRACE) << TXT_BIYLW << "LubyGPU - number of colors: " << colLuby.getColoringGPU()->nCol << TXT_NORML;
	LOG(TRACE) << TXT_BIYLW << "LubyGPU elapsed time: " << duration << TXT_NORML;

	ColoringMCMCParams aa;
	aa.nCol = 70;//test.getMaxNodeDeg() / 2.0f;
	aa.epsilon = 1e-12;
	aa.lambda = 2.0f;
	aa.ratioFreezed = 0.1f;

	//// GPU MCMC coloring

	// ColoringMCMC<float, float> colMCMC(&graph_d, GPURandGen.randStates, aa);
	// start = std::clock();
	// colMCMC.run();
	// duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

	ColoringMCMC_CPU<float, float> mcmc_cpu( &test, aa, seed );
	g_debugger = new dbg( &test, &mcmc_cpu );
	start = std::clock();
	mcmc_cpu.run();
	duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;

	LOG(TRACE) << TXT_BIYLW  << "MCMC_CPU elapsed time: " << duration << TXT_NORML;

	if (g_debugger != nullptr)
		delete g_debugger;

	return EXIT_SUCCESS;
}
